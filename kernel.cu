#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "image_blur.h"
#include "helpers.h"
#include <iostream>
#include <cmath>

__global__ void blur_cuda(unsigned char* input_image, unsigned char* output_image, int width, int height);



void image_blur_cuda(unsigned char* Input_Image, int Height, int Width, int Channels) {
	unsigned char* Dev_Input_Image = NULL;
	unsigned char* Dev_Output_Image = NULL;



	getError(hipMalloc((void**)&Dev_Input_Image, Width * Height * 3 * sizeof(unsigned char)));
	getError(hipMemcpy(Dev_Input_Image, Input_Image, Width * Height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));

	getError(hipMalloc((void**)&Dev_Output_Image, Width * Height * 3 * sizeof(unsigned char)));
    
    
    dim3 blockDims(512, 1, 1);
	dim3 gridDims((unsigned int)ceil((double)(Width * Height * 3 / blockDims.x)), 1, 1);
	

	

	blur_cuda << <gridDims, blockDims >> > (Dev_Input_Image, Dev_Output_Image,Width,Height);

	//copy processed data back to cpu from gpu
	getError(hipMemcpy(Input_Image, Dev_Output_Image, Width * Height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost));

	getError(hipFree(Dev_Input_Image));
	getError(hipFree(Dev_Output_Image));
	//free gpu mempry
	
}


__global__ void blur_cuda(unsigned char* input_image, unsigned char* output_image, int Width, int Height) {
    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int x = offset % Width;
    int y = (offset - x) / Width;
    int fsize = 5; // Filter size
    if (offset < Width * Height) {

        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;
        int hits = 0;
        for (int ox = -fsize; ox < fsize + 1; ++ox) {
            for (int oy = -fsize; oy < fsize + 1; ++oy) {
                if ((x + ox) > -1 && (x + ox) < Width && (y + oy) > -1 && (y + oy) < Height) {
                    const int currentoffset = (offset + ox + oy * Width) * 3;
                    output_red += input_image[currentoffset];
                    output_green += input_image[currentoffset + 1];
                    output_blue += input_image[currentoffset + 2];
                    hits++;
                }
            }
        }
        output_image[offset * 3] = output_red / hits;
        output_image[offset * 3 + 1] = output_green / hits;
        output_image[offset * 3 + 2] = output_blue / hits;
    }
}